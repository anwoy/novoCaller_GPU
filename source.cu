#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#include <iostream>
#include <algorithm>
#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <string>


#include <hipblas.h>
#include <hiprand.h>

#include <boost/numeric/ublas/matrix.hpp>
#include <boost/numeric/ublas/io.hpp>
#include <boost/numeric/ublas/lu.hpp>

#include "basic_operations.h"
#include "matrix_operations.h"
#include "defs01.h"
#include "tests01.h"




int main() {
	tester_01();
	return 0;
}







